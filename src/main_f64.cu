#include "hip/hip_runtime.h"
#include "dasp_f64.h"

int verify_new(MAT_VAL_TYPE *cusp_val, MAT_VAL_TYPE *cuda_val, int *new_order, int length)
{
    for (int i = 0; i < length; i ++)
    {
        int cusp_idx = new_order[i];
        if (fabs(cusp_val[cusp_idx] - cuda_val[i]) > 1e-5)
        {
            printf("error in (%d), cusp(%4.2f), cuda(%4.2f),please check your code!\n", i, cusp_val[cusp_idx], cuda_val[i]);
            return -1;
        }
    }
    printf("Y(%d), compute succeed!\n", length);
    return 0;
}

__host__
void cusparse_spmv_all(MAT_VAL_TYPE *cu_ValA, MAT_PTR_TYPE *cu_RowPtrA, int *cu_ColIdxA, 
                       MAT_VAL_TYPE *cu_ValX, MAT_VAL_TYPE *cu_ValY, int rowA, int colA, MAT_PTR_TYPE nnzA,
                       long long int data_origin1, long long int data_origin2, double *cu_time, double *cu_gflops, double *cu_bandwidth1, double *cu_bandwidth2, double *cu_pre)
{
    struct timeval t1, t2;

    MAT_VAL_TYPE *dA_val, *dX, *dY;
    int *dA_cid;
    MAT_PTR_TYPE *dA_rpt;
    MAT_VAL_TYPE alpha = 1.0, beta = 0.0;

    hipMalloc((void **)&dA_val, sizeof(MAT_VAL_TYPE) * nnzA);
    hipMalloc((void **)&dA_cid, sizeof(int) * nnzA);
    hipMalloc((void **)&dA_rpt, sizeof(MAT_PTR_TYPE) * (rowA + 1));
    hipMalloc((void **)&dX, sizeof(MAT_VAL_TYPE) * colA);
    hipMalloc((void **)&dY, sizeof(MAT_VAL_TYPE) * rowA);

    hipMemcpy(dA_val, cu_ValA, sizeof(MAT_VAL_TYPE) * nnzA, hipMemcpyHostToDevice);
    hipMemcpy(dA_cid, cu_ColIdxA, sizeof(int) * nnzA, hipMemcpyHostToDevice);
    hipMemcpy(dA_rpt, cu_RowPtrA, sizeof(MAT_PTR_TYPE) * (rowA + 1), hipMemcpyHostToDevice);
    hipMemcpy(dX, cu_ValX, sizeof(MAT_VAL_TYPE) * colA, hipMemcpyHostToDevice);
    hipMemset(dY, 0.0, sizeof(MAT_VAL_TYPE) * rowA);

    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer = NULL;
    size_t               bufferSize = 0;

    gettimeofday(&t1, NULL);
    hipsparseCreate(&handle);
    hipsparseCreateCsr(&matA, rowA, colA, nnzA, dA_rpt, dA_cid, dA_val,
                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    hipsparseCreateDnVec(&vecX, colA, dX, HIP_R_64F);
    hipsparseCreateDnVec(&vecY, rowA, dY, HIP_R_64F);
    hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                            HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);
    // hipDeviceSynchronize();
    gettimeofday(&t2, NULL);
    double cusparse_pre = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;
    // printf("cusparse preprocessing time: %8.4lf ms\n", cusparse_pre);
    *cu_pre = cusparse_pre;

    for (int i = 0; i < 100; ++i)
    {
        hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                    HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);
    }
    hipDeviceSynchronize();

    gettimeofday(&t1, NULL);
    for (int i = 0; i < 1000; ++i)
    {
        hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                    HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);
    }
    hipDeviceSynchronize();
    gettimeofday(&t2, NULL);
    *cu_time = ((t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0) / 1000;
    *cu_gflops = (double)((long)nnzA * 2) / (*cu_time * 1e6);
    *cu_bandwidth1 = (double)data_origin1 / (*cu_time * 1e6); 
    *cu_bandwidth2 = (double)data_origin2 / (*cu_time * 1e6); 
    printf("cusparse:%8.4lf ms, %8.4lf Gflop/s, %9.4lf GB/s, %9.4lf GB/s\n", *cu_time, *cu_gflops, *cu_bandwidth1, *cu_bandwidth2);

    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);
    hipsparseDestroy(handle);

    hipMemcpy(cu_ValY, dY, sizeof(MAT_VAL_TYPE) * rowA, hipMemcpyDeviceToHost);

    hipFree(dA_val);
    hipFree(dA_cid);
    hipFree(dA_rpt);
    hipFree(dX);
    hipFree(dY);
}

__host__
int main(int argc, char **argv)
{
    if (argc < 2)
    {
        printf("Run the code by './spmv_double matrix.mtx'. \n");
        return 0;
    }

    // struct timeval t1, t2;
    int rowA, colA;
    MAT_PTR_TYPE nnzA;
    int isSymmetricA;
    MAT_VAL_TYPE *csrValA;
    int *csrColIdxA;
    MAT_PTR_TYPE *csrRowPtrA;

    char *filename;
    filename = argv[1];
    // int NUM = atoi(argv[2]);
    // int block_longest = atoi(argv[3]);
    int NUM = 4;
    int block_longest = 256;
    double threshold = 0.75;

    printf("\n===%s===\n\n", filename);

    mmio_allinone(&rowA, &colA, &nnzA, &isSymmetricA, &csrRowPtrA, &csrColIdxA, &csrValA, filename);
    MAT_VAL_TYPE *X_val = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * colA);
    initVec(X_val, colA);
    initVec(csrValA, nnzA);

    printf("INIT DONE\n");

    MAT_VAL_TYPE *dY_val = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * rowA);
    MAT_VAL_TYPE *Y_val = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * rowA);
    int *new_order = (int *)malloc(sizeof(int) * rowA);

    // int warmup = 3, pre_num = 10;
    
    double cu_time = 0, cu_gflops = 0, cu_bandwidth1 = 0, cu_bandwidth2 = 0, cu_pre = 0;
    long long int data_origin1 = (nnzA + colA + rowA) * sizeof(MAT_VAL_TYPE) + nnzA * sizeof(int) + (rowA + 1) * sizeof(MAT_PTR_TYPE);
    long long int data_origin2 = (nnzA + nnzA + rowA) * sizeof(MAT_VAL_TYPE) + nnzA * sizeof(int) + (rowA + 1) * sizeof(MAT_PTR_TYPE);
    
    cusparse_spmv_all(csrValA, csrRowPtrA, csrColIdxA, X_val, dY_val, rowA, colA, nnzA, data_origin1, data_origin2, &cu_time, &cu_gflops, &cu_bandwidth1, &cu_bandwidth2, &cu_pre);
    
    // double dasp_pre = 0;
    spmv_all(filename, csrValA, csrRowPtrA, csrColIdxA, X_val, Y_val, new_order, rowA, colA, nnzA, NUM, threshold, block_longest);

    FILE* fout;
    fout = fopen("data/spmv_f64_record.csv", "a");
    fprintf(fout, "%lld,%lf,%lf,%lf,%lf\n", data_origin1, cu_time, cu_gflops, cu_bandwidth1, cu_bandwidth2);
    fclose(fout);
    
    /* verify the result with cusparse */
    // int result = verify_new(dY_val, Y_val, new_order, rowA);

    free(X_val);
    free(Y_val);
    free(dY_val);
    free(csrColIdxA);
    free(csrRowPtrA);
    free(csrValA);
    free(new_order);

    return 0;
}